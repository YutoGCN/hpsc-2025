#include <cstdio>
#include <vector>
#include <cstdlib>
#include <hip/hip_runtime.h>

__global__ void count_bucket(const int* key, int* bucket, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        atomicAdd(&bucket[key[i]], 1); 
    }
}

__global__ void bucket_to_key(int* key, const int* bucket, int n, int range) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        int count = 0;
        for (int val = 0; val < range; ++val) {
            count += bucket[val];
            if (i < count) {
                key[i] = val;
                break;
            }
        }
    }
}

int main() {
    int n = 50;
    int range = 5;
    std::vector<int> key(n);
    for (int i=0; i<n; i++) {
        key[i] = rand() % range;
        printf("%d ",key[i]);
    }
    printf("\n");

    int* d_key;
    int* d_bucket;
    hipMalloc(&d_key, n * sizeof(int));
    hipMalloc(&d_bucket, range * sizeof(int));

    hipMemcpy(d_key, key.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_bucket, 0, range * sizeof(int)); 

    int threads = 128;
    int blocks = (n + threads - 1) / threads;
    count_bucket<<<blocks, threads>>>(d_key, d_bucket, n);
    hipDeviceSynchronize();

    bucket_to_key<<<blocks, threads>>>(d_key, d_bucket, n, range);
    hipDeviceSynchronize();

    hipMemcpy(key.data(), d_key, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i=0; i<n; i++) printf("%d ",key[i]);
    printf("\n");

    hipFree(d_key);
    hipFree(d_bucket);
    return 0;
}
